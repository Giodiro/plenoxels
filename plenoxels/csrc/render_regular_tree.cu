#include "hip/hip_runtime.h"
#include <cmath>
#include <stdexcept>
#include <tuple>

#include <torch/torch.h>
#include <torch/extension.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAGuard.h>
#include <cub/warp/warp_reduce.cuh>

#include "render_util.cuh"
#include "cuda_util.cuh"

template <typename T, size_t N>
using Acc32 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int32_t>;
template <typename T, size_t N>
using Acc64 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int64_t>;


const int CUDA_THREADS_PER_BLOCK = 128;
const int WARP_SIZE = 32;
const int CUDA_WARPS_PER_BLOCK = CUDA_THREADS_PER_BLOCK / WARP_SIZE;


constexpr uint32_t n_blocks_linear(uint32_t n_elements, uint32_t n_threads_linear) {
    return (uint32_t)(n_elements + n_threads_linear - 1) / n_threads_linear;
}


__device__ __inline__ int32_t coo2idx(int32_t x, int32_t y, int32_t z, uint32_t grid_size) {
    return x + y * grid_size + z * grid_size * grid_size;
}

__constant__
static const float OFFSET[8][3] = {{-0.5, -0.5, -0.5}, {-0.5, -0.5, 0.5}, {-0.5, 0.5, -0.5}, {-0.5, 0.5, 0.5},
                                   {0.5, -0.5, -0.5},  {0.5, -0.5, 0.5},  {0.5, 0.5, -0.5},  {0.5, 0.5, 0.5}};


template<typename scalar_t>
__device__ __inline__ void dictionary_grad(
    const Acc32<scalar_t, 2> coarse_grid,   // Rc^3, S
    const Acc32<scalar_t, 3> atoms,         // Rf^3, S, D
    Acc32<scalar_t, 2> d_coarse_grid,       // Rc^3, S
    Acc32<scalar_t, 3> d_atoms,             // Rf^3, S, D
    const scalar_t grad_output,             // 1
    const scalar_t * __restrict__ point,    // 3
    const fast_divmod& fast_divmod_fine_reso,
    const hipcub::WarpReduce<scalar_t>& cub_reduce,
    const uint32_t coarse_reso,
    const uint32_t warp_lane
)
{
    const uint32_t tot_reso = coarse_reso * fast_divmod_fine_reso.d_;
    const uint32_t D = atoms.size(2);
    const uint32_t S = coarse_grid.size(1);
    const scalar_t fp[3] = {point[0] * tot_reso, point[1] * tot_reso, point[2] * tot_reso};

    int32_t cn[3];           // corner of coarse-neighbor cell in 'coarse-grid' coordinates
    int32_t fn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates
    int32_t rfn[3];          // corner of fine-neighbor cell in 'full-grid' coordinates

    scalar_t interp_weight;
    int32_t cn_realcoo;
    int32_t fn_realcoo;
    for (int i = 0; i < 8; i++) {
        fn[0] = floor2int(fp[0] + OFFSET[i][0]);
        fn[1] = floor2int(fp[1] + OFFSET[i][1]);
        fn[2] = floor2int(fp[2] + OFFSET[i][2]);
        if (fn[0] < 0 || fn[0] >= tot_reso ||
            fn[1] < 0 || fn[1] >= tot_reso ||
            fn[2] < 0 || fn[2] >= tot_reso) {
            continue;
        }
        fast_divmod_fine_reso.divmod(fn[0], cn[0], rfn[0]);  // fn[0] = fn[0] / fine_reso, cn[0] = fn[0] % fine_reso;
        fast_divmod_fine_reso.divmod(fn[1], cn[1], rfn[1]);
        fast_divmod_fine_reso.divmod(fn[2], cn[2], rfn[2]);
        cn_realcoo = coo2idx(cn[0], cn[1], cn[2], coarse_reso);
        fn_realcoo = coo2idx(rfn[0], rfn[1], rfn[2], fast_divmod_fine_reso.d_);

        interp_weight = (1.0f - myabs(fp[0] - static_cast<scalar_t>(fn[0]) - 0.5f)) *
                        (1.0f - myabs(fp[1] - static_cast<scalar_t>(fn[1]) - 0.5f)) *
                        (1.0f - myabs(fp[2] - static_cast<scalar_t>(fn[2]) - 0.5f));
        interp_weight *= grad_output;

        for (uint32_t s = 0; s < S; s++) {
            atomicAdd(
                &d_atoms[fn_realcoo][s][warp_lane],
                coarse_grid[cn_realcoo][s] * interp_weight);  // TODO: NOT COALESCED.

            scalar_t grad_cg = cub_reduce.Sum(atoms[fn_realcoo][s][warp_lane] * interp_weight, D);
            if (warp_lane == 0) {
                atomicAdd(&d_coarse_grid[cn_realcoo][s], grad_cg);  // TODO: NOT COALESCED.
            }
            __syncwarp((1U << D) - 1);
        }
    }
}


template<typename scalar_t>
__device__ __inline__ void
dictionary_interp(const Acc32<scalar_t, 2> coarse_grid,  // Rc^3, S
                  const Acc32<scalar_t, 3> atoms,        // Rf^3, S, D
                  const scalar_t * __restrict__ point,    // 3
                        scalar_t * __restrict__ coarse_w_smem,  // num warps in block * S
                        scalar_t * __restrict__ out,      // 1
                  const fast_divmod& fast_divmod_fine_reso,
                  const uint32_t coarse_reso,
                  const uint32_t warp_lane,
                  const uint32_t warp_offset)
{
    const uint32_t tot_reso = coarse_reso * fast_divmod_fine_reso.d_;
    const uint32_t D = atoms.size(2);
    const uint32_t S = coarse_grid.size(1);
    const scalar_t fp[3] = {point[0] * tot_reso, point[1] * tot_reso, point[2] * tot_reso};

    int32_t cn[3];           // corner of coarse-neighbor cell in 'coarse-grid' coordinates
    int32_t fn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates
    int32_t rfn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates

    scalar_t interp_weight;
    *out = 0.0f;

    int32_t cn_realcoo;
    int32_t fn_realcoo;
    for (int i = 0; i < 8; i++) {
        fn[0] = floor2int(fp[0] + OFFSET[i][0]);
        fn[1] = floor2int(fp[1] + OFFSET[i][1]);
        fn[2] = floor2int(fp[2] + OFFSET[i][2]);
        if (fn[0] < 0 || fn[0] >= tot_reso ||
            fn[1] < 0 || fn[1] >= tot_reso ||
            fn[2] < 0 || fn[2] >= tot_reso) {
            continue;
        }
        fast_divmod_fine_reso.divmod(fn[0], cn[0], rfn[0]);  // fn[0] = fn[0] / fine_reso, cn[0] = fn[0] % fine_reso;
        fast_divmod_fine_reso.divmod(fn[1], cn[1], rfn[1]);
        fast_divmod_fine_reso.divmod(fn[2], cn[2], rfn[2]);
        cn_realcoo = coo2idx(cn[0], cn[1], cn[2], coarse_reso);
        fn_realcoo = coo2idx(rfn[0], rfn[1], rfn[2], fast_divmod_fine_reso.d_);

        interp_weight = (1.0f - myabs(fp[0] - static_cast<scalar_t>(fn[0]) - 0.5f)) *
                        (1.0f - myabs(fp[1] - static_cast<scalar_t>(fn[1]) - 0.5f)) *
                        (1.0f - myabs(fp[2] - static_cast<scalar_t>(fn[2]) - 0.5f));
        // load w from coarse_grid to shared mem using all active threads in warp
        for (int s = warp_lane; s < S; s += D) {
            coarse_w_smem[warp_offset * S + s] = coarse_grid[cn_realcoo][s];
        }
        __syncwarp((1U << D) - 1);
        for (int s = 0; s < S; s++) {
            // pseudo: out += coarse_grid[cn][s] * iw[j] * atoms[fn][s][d]
            myfma(coarse_w_smem[warp_offset * S + s] * interp_weight,
                  atoms[fn_realcoo][s][warp_lane],
                  out);
        }
        __syncwarp((1U << D) - 1);
    }
}



template<typename scalar_t, uint32_t BASIS_DIM>
__global__ void
trace_ray(
    const Acc32<scalar_t, 2> coarse_grid,
    const Acc32<scalar_t, 3> atoms,
    const Acc32<scalar_t, 2> rays_o,
    const Acc32<scalar_t, 2> rays_d,
    Acc32<scalar_t, 2> out,
    const fast_divmod fast_divmod_fine_reso,
    const uint32_t coarse_reso,
    const uint32_t n_rays,
    const scalar_t * __restrict__ scaling,
    const scalar_t * __restrict__ offset,
    const RenderOptions opt
)
{
    const uint32_t ray_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    const uint32_t warp_offset = threadIdx.x / WARP_SIZE;
    const uint32_t warp_lane = threadIdx.x % WARP_SIZE;
    const uint32_t lane_colorgrp = warp_lane / BASIS_DIM;
    const uint32_t lane_colorgrp_id = warp_lane % BASIS_DIM;
    const uint32_t D = atoms.size(2);  // D is also BASIS_DIM * 3 + 1
    typedef hipcub::WarpReduce<scalar_t> WarpReduce;
	if (ray_id >= n_rays || warp_lane >= D) return;

    // shared memory. This is done to save some register space, no actual sharing occurs.
    __shared__ scalar_t sphfunc_val[CUDA_WARPS_PER_BLOCK][9];
    __shared__ Ray<scalar_t> ray_spec[CUDA_WARPS_PER_BLOCK];
    __shared__ typename WarpReduce::TempStorage cub_storage[CUDA_WARPS_PER_BLOCK];
    // dynamically allocated shmem. This is actually shared
    scalar_t* coarse_w_smem = shared_memory_proxy<scalar_t>();  // CUDA_WARPS_PER_BLOCK * S

    // Setup the ray-spec. Will copy data from rays_o, rays_d
    ray_spec[warp_offset].set(rays_o[ray_id].data(), rays_d[ray_id].data());
    // Spherical harmonics are computed before ray normalization
    calc_sphfunc(/*basis_dim=*/BASIS_DIM, /*dir=*/ray_spec[warp_offset].dir, /*out=*/sphfunc_val[warp_offset]);
    // Finish ray-spec initialization
    ray_find_bounds(ray_spec[warp_offset], scaling, offset, (scalar_t)opt.step_size, (scalar_t)opt.near_plane);
    __syncwarp((1U << D) - 1);

    if (ray_spec[warp_offset].tmin > ray_spec[warp_offset].tmax) {  // Ray doesn't hit box
        out[ray_id][lane_colorgrp] = 1.0f;
        return;
    }

    scalar_t t = ray_spec[warp_offset].tmin;
    scalar_t outv = 0.0f;
    scalar_t log_light_intensity = 0.0f;
    scalar_t sigma, interp_val;
    while (t <= ray_spec[warp_offset].tmax) {
        ray_spec[warp_offset].update_pos(t);

        dictionary_interp(
            coarse_grid, atoms, /*point=*/ray_spec[warp_offset].pos, /*coarse_w_smem=*/coarse_w_smem,
            /*out=*/&interp_val, fast_divmod_fine_reso, coarse_reso, warp_lane, warp_offset);
        sigma = interp_val;  // This has an effect only in last thread in active warp.
        // broadcast sigma (stored in last coordinate) to other threads in warp
        sigma = __shfl_sync((1U << D) - 1, sigma, /*srcLane=*/D - 1);
        if (sigma > opt.sigma_thresh) {
            interp_val *= sphfunc_val[warp_offset][lane_colorgrp_id]; // bank conflict
            const scalar_t pcnt = ray_spec[warp_offset].world_step * sigma;
            const scalar_t weight = myexp(log_light_intensity) * (1.f - myexp(-pcnt));
            log_light_intensity -= pcnt;

            // The reduction will also happen on the last lane which only holds sigma.
            // The value computed there is ignored.
            scalar_t lane_color_total = WarpReduce(cub_storage[warp_offset]).HeadSegmentedSum(
                interp_val, lane_colorgrp_id == 0);
            outv += weight * mymax(lane_color_total + 0.5f, 0.0f);  // clamp [+0, infty)
            if (myexp(log_light_intensity) < opt.stop_thresh) {
                log_light_intensity = -1e3f;
                break;
            }
        }
        t += opt.step_size;
    }
    outv += myexp(log_light_intensity) * 1.0f;
    if (lane_colorgrp_id == 0 && lane_colorgrp < 3) {
        out[ray_id][lane_colorgrp] = outv;
    }
}



template <typename scalar_t, uint32_t BASIS_DIM>
__device__ __inline__
void trace_ray_cuvol_backward(
        const Acc32<scalar_t, 2> grad_output,  // N, 3
        const Acc32<scalar_t, 2> color_cache,  // N, 3
        const Acc32<scalar_t, 2> coarse_grid,
        const Acc32<scalar_t, 3> atoms,
        const Acc32<scalar_t, 2> rays_o,
        const Acc32<scalar_t, 2> rays_d,
        Acc32<scalar_t, 2> d_coarse_grid,
        Acc32<scalar_t, 3> d_atoms,
        const fast_divmod fast_divmod_fine_reso,
        const uint32_t coarse_reso,
        const uint32_t n_rays,
        const scalar_t * __restrict__ scaling,
        const scalar_t * __restrict__ offset,
        const RenderOptions opt
)
{
    const uint32_t ray_id = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    const uint32_t warp_offset = threadIdx.x / WARP_SIZE;
    const uint32_t warp_lane = threadIdx.x % WARP_SIZE;
    const uint32_t lane_colorgrp = warp_lane / BASIS_DIM;
    const uint32_t lane_colorgrp_id = warp_lane % BASIS_DIM;
    // if BASIS_DIM=9, leader_mask=0000 1000 0000 0100 0000 0010 0000 0001 selecting the leaders of the color
    // groups and the final sigma dimension
    const uint32_t leader_mask = 1U | (1U << BASIS_DIM) | (1U << (2 * BASIS_DIM)) | (1U << (3 * BASIS_DIM));
    const uint32_t D = atoms.size(2);  // D is also BASIS_DIM * 3 + 1
    typedef hipcub::WarpReduce<scalar_t> WarpReduce;
	if (ray_id >= n_rays || warp_lane >= D) return;

    // shared memory. This is done to save some register space, no actual sharing occurs.
    __shared__ scalar_t sphfunc_val[CUDA_WARPS_PER_BLOCK][9];
    __shared__ Ray<scalar_t> ray_spec[CUDA_WARPS_PER_BLOCK];
    __shared__ typename WarpReduce::TempStorage cub_storage[CUDA_WARPS_PER_BLOCK];
    // dynamically allocated shmem. This is actually shared
    scalar_t* coarse_w_smem = shared_memory_proxy<scalar_t>();  // CUDA_WARPS_PER_BLOCK * S

    // Setup the ray-spec. Will copy data from rays_o, rays_d
    ray_spec[warp_offset].set(rays_o[ray_id].data(), rays_d[ray_id].data());
    // Spherical harmonics are computed before ray normalization
    calc_sphfunc(/*basis_dim=*/BASIS_DIM, /*dir=*/ray_spec[warp_offset].dir, /*out=*/sphfunc_val[warp_offset]);
    // Finish ray-spec initialization
    ray_find_bounds(ray_spec[warp_offset], scaling, offset, (scalar_t)opt.step_size, (scalar_t)opt.near_plane);

    scalar_t c_grad_out[3];
    const scalar_t norm_factor = 2.0f / (3 * n_rays);
    #pragma unroll 3
    for (int i = 0; i < 3; ++i) {
        c_grad_out[i] = (color_cache[ray_id][i] - grad_output[ray_id][i]) * norm_factor;
    }
    scalar_t accum = fmaf(color_cache[ray_id][0], c_grad_out[0],
                      fmaf(color_cache[ray_id][1], c_grad_out[1],
                           color_cache[ray_id][2] * c_grad_out[2]));

    if (ray_spec[warp_offset].tmin > ray_spec[warp_offset].tmax) {
        return;
    }

    scalar_t t = ray_spec[warp_offset].tmin;
    const scalar_t gout = lane_colorgrp < 3 ? c_grad_out[lane_colorgrp] : 0.0f;  // avoid out-of-bounds on sigma thread
    scalar_t log_light_intensity = 0.0f;
    scalar_t sigma, interp_val;

    // remat samples
    while (t <= ray_spec[warp_offset].tmax) {
        ray_spec[warp_offset].update_pos(t);

        dictionary_interp(
            coarse_grid, atoms, /*point=*/ray_spec[warp_offset].pos, /*coarse_w_smem=*/coarse_w_smem,
            /*out=*/&interp_val, fast_divmod_fine_reso, coarse_reso, warp_lane, warp_offset);
        sigma = interp_val;  // This has an effect only in last thread in active warp.
        // broadcast sigma (stored in last coordinate) to other threads in warp
        sigma = __shfl_sync((1U << D) - 1, sigma, /*srcLane=*/D - 1);

        if (opt.last_sample_opaque && t + opt.step_size > ray_spec[warp_offset].tmax) {
            ray_spec[warp_offset].world_step = 1e9;
        }
        if (sigma > opt.sigma_thresh) {
            scalar_t weighted_lane_color = interp_val * sphfunc_val[warp_offset][lane_colorgrp_id];
            const scalar_t pcnt = ray_spec[warp_offset].world_step * sigma;
            const scalar_t weight = myexp(log_light_intensity) * (1.f - myexp(-pcnt));
            log_light_intensity -= pcnt;

            // Sum over all dimensions for the color of lane_colorgrp_id. Only valid in the head.
            const scalar_t lane_color_total = WarpReduce(cub_storage[warp_offset]).HeadSegmentedSum(
                weighted_lane_color, lane_colorgrp_id == 0) + 0.5f;
            scalar_t total_color = mymax(lane_color_total, 0.0f);  // Clamp to [+0, infty)
            scalar_t color_in_01 = total_color == lane_color_total;
            total_color *= gout;  // the multiplication zeroes out total_color for the sigma lane

            // For each 'leader' thread (first thread in a colorgroup), sum the values in the other leaders.
            scalar_t total_color_c1 = __shfl_sync(leader_mask, total_color, /*srcLane=*/BASIS_DIM);
            total_color += __shfl_sync(leader_mask, total_color, 2 * BASIS_DIM);
            total_color += total_color_c1;

            // for sigma thread this will be something random
            color_in_01 = __shfl_sync((1U << D) - 1, color_in_01, /*srcLane=*/lane_colorgrp * BASIS_DIM);
            const scalar_t grad_common = weight * color_in_01 * gout;
            const scalar_t curr_grad_color = sphfunc_val[warp_offset][lane_colorgrp_id] * grad_common;

            accum -= weight * total_color;
            scalar_t curr_grad_sigma = ray_spec[warp_offset].world_step * (total_color * myexp(log_light_intensity) - accum);

            if (warp_lane == D - 1) {
                dictionary_grad(coarse_grid, atoms, d_coarse_grid, d_atoms, /*grad_output=*/curr_grad_sigma,
                                /*point=*/ray_spec[warp_offset].pos, fast_divmod_fine_reso,
                                WarpReduce(cub_storage[warp_offset]), coarse_reso, warp_lane);
            } else {
                dictionary_grad(coarse_grid, atoms, d_coarse_grid, d_atoms, /*grad_output=*/curr_grad_color,
                                /*point=*/ray_spec[warp_offset].pos, fast_divmod_fine_reso,
                                WarpReduce(cub_storage[warp_offset]), coarse_reso, warp_lane);
            }
            if (myexp(log_light_intensity) < opt.stop_thresh) {
                break;
            }
        }
        t += opt.step_size;
    }
}







using torch::autograd::variable_list;
using torch::autograd::tensor_list;
using torch::autograd::Function;
using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::Tensor;


class DictTreeRender : public Function<DictTreeRender> {
    public:
        static Tensor forward(AutogradContext *ctx,
                              Tensor coarse_grid,   // Rc^3, S
                              Tensor atoms,         // Rf^3, S, D
                              Tensor rays_o,        // N, 3
                              Tensor rays_d,        // N, 3
                              int64_t fine_reso,
                              int64_t coarse_reso,
                              double scaling,
                              double offset,
                              double step_size,
                              double sigma_thresh,
                              double stop_thresh)
        {
            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();
            // Size checks
            if (coarse_grid.size(0) != coarse_reso * coarse_reso * coarse_reso) {
                throw std::invalid_argument("Coarse-grid has wrong first dimension");
            }
            if (coarse_grid.size(1) != atoms.size(1)) {
                throw std::invalid_argument("Coarse-grid and atoms dimension 1 doesn't match");
            }
            if (atoms.size(0) != fine_reso * fine_reso * fine_reso) {
                throw std::invalid_argument("Atoms has wrong first dimension");
            }
            if (atoms.size(2) > 32) {
                throw std::invalid_argument("Data dimension must be at most 32");
            }
            RenderOptions opt = {
                .step_size = (float)step_size,
                .sigma_thresh = (float)sigma_thresh,
                .stop_thresh = (float)stop_thresh,
                .near_plane = 0.0f,
                .last_sample_opaque = true
            };

            const uint32_t num_rays = rays_o.size(0);

            auto out = torch::zeros({num_rays, 3}, coarse_grid.options());
            auto scaling_t = torch::tensor({scaling, scaling, scaling}, coarse_grid.options());
            auto offset_t = torch::tensor({offset, offset, offset}, coarse_grid.options());

            const dim3 grid_size(n_blocks_linear(num_rays, CUDA_WARPS_PER_BLOCK));
            const dim3 block_size(CUDA_THREADS_PER_BLOCK);
            const uint32_t shared_mem = CUDA_WARPS_PER_BLOCK * coarse_grid.size(1);

            fast_divmod fast_divmod_fine_reso((int32_t)fine_reso);

            AT_DISPATCH_FLOATING_TYPES(coarse_grid.scalar_type(), "trace_ray", [&] {
                trace_ray<scalar_t, 1><<<grid_size, block_size, shared_mem * sizeof(scalar_t), stream.stream()>>>(
                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    rays_o.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    rays_d.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    fast_divmod_fine_reso,
                    (uint32_t)coarse_reso,
                    num_rays,
                    scaling_t.data_ptr<scalar_t>(),
                    offset_t.data_ptr<scalar_t>(),
                    opt
                );
            });
            ctx->save_for_backward({coarse_grid, atoms, out});
            ctx->saved_data["rays_o"] = rays_o;
            ctx->saved_data["rays_d"] = rays_o;
            ctx->saved_data["fine_reso"] = fine_reso;
            ctx->saved_data["coarse_reso"] = coarse_reso;
            ctx->saved_data["scaling"] = scaling;
            ctx->saved_data["offset"] = offset;
            ctx->saved_data["step_size"] = step_size;
            ctx->saved_data["sigma_thresh"] = sigma_thresh;
            ctx->saved_data["stop_thresh"] = stop_thresh;
            return out;
        }

        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {
            const auto saved = ctx->get_saved_variables();
            const Tensor coarse_grid = saved[0];
            const Tensor atoms = saved[1];
            const Tensor fwd_output = saved[2];

            const Tensor rays_o = ctx->saved_data["rays_o"].toTensor();
            const Tensor rays_d = ctx->saved_data["rays_d"].toTensor();
            const int64_t coarse_reso = ctx->saved_data["coarse_reso"].toInt();
            const int64_t fine_reso = ctx->saved_data["fine_reso"].toInt();
            const double scaling = ctx->saved_data["scaling"].toDouble();
            const double offset = ctx->saved_data["offset"].toDouble();
            const RenderOptions opt = {
                .step_size = (float)ctx->saved_data["step_size"].toDouble(),
                .sigma_thresh = (float)ctx->saved_data["sigma_thresh"].toDouble(),
                .stop_thresh = (float)ctx->saved_data["stop_thresh"].toDouble(),
                .near_plane = 0.0f,
                .last_sample_opaque = true
            };
            fast_divmod fast_divmod_fine_reso((int32_t)fine_reso);

            const Tensor grad_output = grad_outputs[0];

            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();

            Tensor d_coarse_grid = torch::zeros_like(coarse_grid);
            Tensor d_atoms = torch::zeros_like(atoms);
            auto scaling_t = torch::tensor({scaling, scaling, scaling}, coarse_grid.options());
            auto offset_t = torch::tensor({offset, offset, offset}, coarse_grid.options());

            const dim3 grid_size(n_blocks_linear(rays_o.size(0), CUDA_WARPS_PER_BLOCK));
            const dim3 block_size(CUDA_THREADS_PER_BLOCK);
            const uint32_t shared_mem = CUDA_WARPS_PER_BLOCK * coarse_grid.size(1);

            AT_DISPATCH_FLOATING_TYPES(coarse_grid.scalar_type(), "trace_ray_cuvol_bwd", [&] {
                trace_ray_cuvol_backward<scalar_t, 1><<<grid_size, block_size, shared_mem * sizeof(scalar_t), stream.stream()>>>(
                    grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    fwd_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    rays_o.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    rays_d.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    d_coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    d_atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                    fast_divmod_fine_reso,
                    (uint32_t)coarse_reso,
                    (uint32_t)rays_o.size(0),
                    scaling_t.data_ptr<scalar_t>(),
                    offset_t.data_ptr<scalar_t>(),
                    opt
                );
            });
            return {d_coarse_grid, d_atoms, Tensor(), Tensor(), Tensor(), Tensor(), Tensor(), Tensor(), Tensor(), Tensor(), Tensor()};
        }
};


Tensor dict_tree_render(const Tensor &coarse_grid, const Tensor &atoms, const Tensor &rays_o, const Tensor &rays_d,
                        const int64_t fine_reso, const int64_t coarse_reso, const double scaling, const double offset,
                        const double step_size, const double sigma_thresh, const double stop_thresh)
{
    return DictTreeRender::apply(coarse_grid, atoms, rays_o, rays_d, fine_reso, coarse_reso, scaling, offset,
                                 step_size, sigma_thresh, stop_thresh);
}

static auto registry = torch::RegisterOperators()
                        .op("plenoxels::dict_tree_render", &dict_tree_render);

