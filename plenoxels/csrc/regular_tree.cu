#include "hip/hip_runtime.h"
#include <cmath>
#include <stdexcept>
#include <tuple>

#include <torch/torch.h>
#include <torch/extension.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAGuard.h>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_reduce.cuh>

#include "cuda_util.cuh"


template <typename T, size_t N>
using Acc32 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int32_t>;
template <typename T, size_t N>
using Acc64 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int64_t>;

const int CUDA_THREADS_PER_BLOCK = 512;
const int WARP_SIZE = 32;
const int CUDA_WARPS_PER_BLOCK = CUDA_THREADS_PER_BLOCK / WARP_SIZE;


constexpr uint32_t n_blocks_linear(uint32_t n_elements, uint32_t n_threads_linear) {
    return (uint32_t)(n_elements + n_threads_linear - 1) / n_threads_linear;
}


__device__ __inline__ int32_t coo2idx(int32_t x, int32_t y, int32_t z, uint32_t grid_size) {
    return x + y * grid_size + z * grid_size * grid_size;
}

__constant__
static const float OFFSET[8][3] = {{-0.5, -0.5, -0.5}, {-0.5, -0.5, 0.5}, {-0.5, 0.5, -0.5}, {-0.5, 0.5, 0.5},
                                   {0.5, -0.5, -0.5},  {0.5, -0.5, 0.5},  {0.5, 0.5, -0.5},  {0.5, 0.5, 0.5}};


#define FWD_BLOCK_SIZE_X 32
#define FWD_BLOCK_SIZE_Y 16
#define NUM_POINTS_PER_THREAD 16
#define INNER_POINTS_PER_THREAD 4

#define V1_FWD_BLOCK_SIZE = 512;
#define V1_WARPS_PER_BLOCK = V1_FWD_BLOCK_SIZE / 32;


template<class scalar_t, int32_t S, int32_t POW2_RF>
__global__ void
__launch_bounds__(CUDA_THREADS_PER_BLOCK)
k_l2_interp_v1(Acc32<scalar_t, 2> coarse_grid,  // Rc^3, S
               Acc32<scalar_t, 3> atoms,        // Rf^3, S, D
               Acc32<scalar_t, 2> points,       // N, 3
               Acc32<scalar_t, 2> out,          // N, D
               const uint32_t coarse_reso)
{
    constexpr int32_t fine_reso = 2 << (POW2_RF - 1);
    const int32_t point_id = (blockIdx.x * blockDim.x + threadIdx.x) >> 5;
    const int32_t warp_lane = threadIdx.x % 0x1F;
    const int32_t warp_offset = (threadIdx.x >> 5) * S;  // warp_block_id * S

    const int32_t D = atoms.size(2);
    const uint32_t warp_mask = __ballot_sync(0xffffffff, warp_lane < D);
    if (warp_lane >= D || point_id >= points.size(0)) { return; }

    __shared__ scalar_t coarse_w[V1_WARPS_PER_BLOCK * S]
    const scalar_t fp[3] = {points[point_id][0] * coarse_reso * fine_reso,
                            points[point_id][1] * coarse_reso * fine_reso,
                            points[point_id][2] * coarse_reso * fine_reso};

    int32_t cn[3];           // corner of coarse-neighbor cell in 'coarse-grid' coordinates
    int32_t fn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates
    int32_t rfn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates

    scalar_t acc = 0.0f;
    scalar_t interpolation_weight;

    int32_t cn_realcoo, fn_realcoo;
    for (int i = 0; i < 8; i++) {
        fn[0] = clamp(floor2int(fp[0] + OFFSET[i][0]), 0, fine_reso * coarse_reso - 1);
        fast_divmod_pow2<POW2_RF>(fn[0], cn[0], rfn[0]);

        fn[1] = clamp(floor2int(fp[1] + OFFSET[i][1]), 0, fine_reso * coarse_reso - 1);
        fast_divmod_pow2<POW2_RF>(fn[1], cn[1], rfn[1]);

        fn[2] = clamp(floor2int(fp[2] + OFFSET[i][2]), 0, fine_reso * coarse_reso - 1);
        fast_divmod_pow2<POW2_RF>(fn[2], cn[2], rfn[2]);

        cn_realcoo = coo2idx(cn[0], cn[1], cn[2], coarse_reso);
        fn_realcoo = coo2idx(rfn[0], rfn[1], rfn[2], fine_reso);

        interpolation_weight = (1.0f - myabs(fp[0] - static_cast<scalar_t>(fn[0]) - 0.5f)) *
                               (1.0f - myabs(fp[1] - static_cast<scalar_t>(fn[1]) - 0.5f)) *
                               (1.0f - myabs(fp[2] - static_cast<scalar_t>(fn[2]) - 0.5f));
        // load w from coarse_grid to shared mem using all active threads in warp
        for (int s = warp_lane; s < S; s += D) {
            coarse_w[warp_offset + s] = coarse_grid[cn_realcoo][s];
        }
        __syncwarp(warp_mask);
        for (int s = 0; s < S; s++) {
            // pseudo: out += coarse_grid[cn][s] * iw[j] * atoms[fn][s][d]
            acc = myfma(coarse_w[warp_offset + s] * interpolation_weight, atoms[fn_realcoo][s][warp_lane], acc);
        }
        __syncwarp(warp_mask);
    }
    out[point_id][warp_lane] = acc;
}



template<class scalar_t, int32_t S, int32_t POW2_RF>
__global__ void
__launch_bounds__(FWD_BLOCK_SIZE_X * FWD_BLOCK_SIZE_Y, 2)
k_l2_interp_v2(Acc32<scalar_t, 2> coarse_grid,  // Rc^3, S
               Acc32<scalar_t, 3> atoms,        // Rf^3, D, S
               Acc32<scalar_t, 2> points,       // N, 3
               Acc32<scalar_t, 2> out,          // D, N
               const int32_t coarse_reso)
{
    /*
     * Thread-blocks are of size FWD_BLOCK_SIZE_X, FWD_BLOCK_SIZE_Y
     * The block will handle only FWD_BLOCK_SIZE_Y different fine-resolution points,
     * loading them all cooperatively into a_sh.
     *
     * The thread-block x-dimension is used to subdivide on the S dimension (num-atoms), to coalesce global loads.
     * Each row handles NUM_POINTS_PER_THREAD points in a loop. 
     *
     * The grid is of size:
     *  - num_points / FWD_BLOCK_SIZE_Y / NUM_POINTS_PER_THREAD
     *  - D
     *  - fine_reso^3 / FWD_BLOCK_SIZE_Y  TODO: This should be rounded up
     */
    constexpr int32_t num_s_per_thread = S / FWD_BLOCK_SIZE_X;
    constexpr int32_t fine_reso = 2 << (POW2_RF - 1);
    constexpr int32_t max_rf = fine_reso * fine_reso * fine_reso;
    typedef hipcub::WarpReduce<scalar_t> WarpReduce;

    const int32_t start_point_id = blockIdx.x * FWD_BLOCK_SIZE_Y * NUM_POINTS_PER_THREAD + threadIdx.y * NUM_POINTS_PER_THREAD;

    const int32_t rf_id = min(blockIdx.z * FWD_BLOCK_SIZE_Y + threadIdx.y, max_rf);
    const int32_t num_valid_rf_positions = min(FWD_BLOCK_SIZE_Y, max_rf - blockIdx.z * FWD_BLOCK_SIZE_Y);
    const int32_t dim_id = blockIdx.y;
    const int32_t s_idx = threadIdx.x;

    int32_t cn[3];           // corner of coarse-neighbor cell in 'coarse-grid' coordinates
    int32_t fn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates
    int32_t rfn[3];          // corner of fine-neighbor cell in 'full-grid' coordinates

    __shared__ scalar_t a_sh[FWD_BLOCK_SIZE_Y][S];
    __shared__ typename WarpReduce::TempStorage cub_storage[FWD_BLOCK_SIZE_Y];

    scalar_t accs[INNER_POINTS_PER_THREAD];
    scalar_t interpolation_weight;
    scalar_t cg_reg;

    for (int s = 0; s < num_s_per_thread; s++) {
        if (s * FWD_BLOCK_SIZE_X + s_idx < S) {
            a_sh[threadIdx.y][s * FWD_BLOCK_SIZE_X + s_idx] = atoms[rf_id][dim_id][s * FWD_BLOCK_SIZE_X + s_idx];
        } else {
            a_sh[threadIdx.y][s * FWD_BLOCK_SIZE_X + s_idx] = 0.0f;
        }
    }
    __syncthreads();

    int32_t cn_realcoo, fn_realcoo;
    for (int p = 0; p < NUM_POINTS_PER_THREAD; p += INNER_POINTS_PER_THREAD) {
        int j = 0;
        for (; j < INNER_POINTS_PER_THREAD; j++) {
            int point_id = start_point_id + p + j;
            if (point_id >= out.size(1)) break;
            accs[j] = 0.0f;
            scalar_t fp[3] = {points[point_id][0] * fine_reso * coarse_reso, points[point_id][1] * fine_reso * coarse_reso, points[point_id][2] * fine_reso * coarse_reso};
            for (int i = 0; i < 8; i++) {
                fn[0] = clamp(floor2int(fp[0] + OFFSET[i][0]), 0, fine_reso * coarse_reso - 1);
                fast_divmod_pow2<POW2_RF>(fn[0], cn[0], rfn[0]);

                fn[1] = clamp(floor2int(fp[1] + OFFSET[i][1]), 0, fine_reso * coarse_reso - 1);
                fast_divmod_pow2<POW2_RF>(fn[1], cn[1], rfn[1]);

                fn[2] = clamp(floor2int(fp[2] + OFFSET[i][2]), 0, fine_reso * coarse_reso - 1);
                fast_divmod_pow2<POW2_RF>(fn[2], cn[2], rfn[2]);

                cn_realcoo = coo2idx(cn[0], cn[1], cn[2], coarse_reso);
                fn_realcoo = coo2idx(rfn[0], rfn[1], rfn[2], fine_reso) - blockIdx.z * FWD_BLOCK_SIZE_Y;

                if (fn_realcoo >= 0 && fn_realcoo < num_valid_rf_positions) {
                    interpolation_weight = (1.0f - myabs(fp[0] - static_cast<scalar_t>(fn[0]) - 0.5f)) *
                                           (1.0f - myabs(fp[1] - static_cast<scalar_t>(fn[1]) - 0.5f)) *
                                           (1.0f - myabs(fp[2] - static_cast<scalar_t>(fn[2]) - 0.5f));
                    for (int s = 0; s < num_s_per_thread; s++) {
                        // out-of-bounds reads will be zeroed out when multiplied by a_sh
                        cg_reg = coarse_grid[cn_realcoo][min(s * FWD_BLOCK_SIZE_X + s_idx, S)]; 
                        cg_reg *= interpolation_weight;
                        accs[j] = myfma(cg_reg, a_sh[fn_realcoo][s * FWD_BLOCK_SIZE_X + s_idx], accs[j]);
                    }
                }
            }
            accs[j] = WarpReduce(cub_storage[threadIdx.y]).Sum(accs[j]);
        }
        if (s_idx == 0) {
            for (int k = 0; k < j; k++) {
                atomicAdd(&out[dim_id][start_point_id + p + k], accs[k]);
            }
        }
    }
}




using torch::autograd::variable_list;
using torch::autograd::tensor_list;
using torch::autograd::Function;
using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::Tensor;


class L2InterpFunctionv1 : public Function<L2InterpFunctionv1> {
    public:
        static Tensor forward(AutogradContext *ctx,
                              Tensor coarse_grid,   // Rc^3, S
                              Tensor atoms,         // Rf^3, S, D
                              Tensor points,        // N, 3
                              int64_t fine_reso,
                              int64_t coarse_reso)
        {
            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();
            // Size checks
            if (coarse_grid.size(0) != coarse_reso * coarse_reso * coarse_reso) {
                throw std::invalid_argument("Coarse-grid has wrong first dimension");
            }
            if (coarse_grid.size(1) != atoms.size(2)) {
                throw std::invalid_argument("Coarse-grid and atoms dimension 1 doesn't match");
            }
            if (atoms.size(0) != fine_reso * fine_reso * fine_reso) {
                throw std::invalid_argument("Atoms has wrong first dimension");
            }
            if (atoms.size(1) > 32) {
                throw std::invalid_argument("Data dimension must be at most 32");
            }

            ctx->save_for_backward({coarse_grid, atoms});
            ctx->saved_data["points"] = points;
            ctx->saved_data["fine_reso"] = fine_reso;
            ctx->saved_data["coarse_reso"] = coarse_reso;

            const int64_t D = atoms.size(2);
            const int64_t S = atoms.size(1);
            auto out = torch::zeros({points.size(0), D}, atoms.options());

            const dim3 grid_size(n_blocks_linear(points.size(0), V1_WARPS_PER_BLOCK));
            const dim3 block_size(V1_FWD_BLOCK_SIZE);

            fast_divmod fast_divmod_fine_reso((int32_t)fine_reso);
            AT_DISPATCH_FLOATING_TYPES(coarse_grid.scalar_type(), "dispatch_l2interpv1_fwd", [&] {
                switch(fine_reso) {
                    case 4:
                        switch (S) {
                            case 64:
                                k_l2_interp_v1<scalar_t, 64, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v1<scalar_t, 128, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v1<scalar_t, 256, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                    case 8:
                        switch (S) {
                            case 64:
                                k_l2_interp_v1<scalar_t, 64, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v1<scalar_t, 128, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v1<scalar_t, 256, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                }
            });
            return out;
        }

        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {
            const auto saved = ctx->get_saved_variables();
            const Tensor coarse_grid = saved[0];
            const Tensor atoms = saved[1];

            const Tensor points = ctx->saved_data["points"].toTensor();
            const int64_t coarse_reso = ctx->saved_data["coarse_reso"].toInt();
            const int64_t fine_reso = ctx->saved_data["fine_reso"].toInt();

            const Tensor grad_output = grad_outputs[0];

            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();

            Tensor d_coarse_grid = torch::zeros_like(coarse_grid);
            Tensor d_atoms = torch::zeros_like(atoms);

            const dim3 grid_size_dcg(n_blocks_linear(points.size(0), CUDA_THREADS_PER_BLOCK / WARP_SIZE));
            const dim3 block_size_dcg(CUDA_THREADS_PER_BLOCK);

            const dim3 grid_size_da(points.size(0));
            const dim3 block_size_da(round_up(grad_output.size(1), 32), 8);  // D * S

            return {d_coarse_grid, d_atoms, Tensor(), Tensor(), Tensor(), Tensor(), Tensor()};
        }
};


class L2InterpFunctionv2 : public Function<L2InterpFunctionv2> {
    public:
        static Tensor forward(AutogradContext *ctx,
                              Tensor coarse_grid,   // Rc^3, S
                              Tensor atoms,         // Rf^3, D, S
                              Tensor points,        // N, 3
                              int64_t fine_reso,
                              int64_t coarse_reso)
        {
            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();
            // Size checks
            if (coarse_grid.size(0) != coarse_reso * coarse_reso * coarse_reso) {
                throw std::invalid_argument("Coarse-grid has wrong first dimension");
            }
            if (coarse_grid.size(1) != atoms.size(2)) {
                throw std::invalid_argument("Coarse-grid and atoms dimension 1 doesn't match");
            }
            if (atoms.size(0) != fine_reso * fine_reso * fine_reso) {
                throw std::invalid_argument("Atoms has wrong first dimension");
            }
//            if (atoms.size(1) > 32) {
//                throw std::invalid_argument("Data dimension must be at most 32");
//            }

            ctx->save_for_backward({coarse_grid, atoms});
            ctx->saved_data["points"] = points;
            ctx->saved_data["fine_reso"] = fine_reso;
            ctx->saved_data["coarse_reso"] = coarse_reso;

            const int64_t D = atoms.size(1);
            const int64_t S = atoms.size(2);
            auto out = torch::zeros({D, points.size(0)}, atoms.options());

            const dim3 grid_size((uint32_t)(points.size(0) / FWD_BLOCK_SIZE_Y / NUM_POINTS_PER_THREAD), 
                                 (uint32_t)D, 
                                 (uint32_t)(fine_reso * fine_reso * fine_reso) / FWD_BLOCK_SIZE_Y);
            const dim3 block_size(FWD_BLOCK_SIZE_X, FWD_BLOCK_SIZE_Y);
//            const dim3 grid_size(n_blocks_linear(points.size(0), CUDA_WARPS_PER_BLOCK));
//            const dim3 block_size(CUDA_THREADS_PER_BLOCK);
//            const uint32_t shared_mem = CUDA_WARPS_PER_BLOCK * coarse_grid.size(1);

            fast_divmod fast_divmod_fine_reso((int32_t)fine_reso);
            AT_DISPATCH_FLOATING_TYPES(coarse_grid.scalar_type(), "dispatch_l2interpv2_fwd", [&] {
                switch(fine_reso) {
                    case 4:
                        switch (S) {
                            case 64:
                                k_l2_interp_v2<scalar_t, 64, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v2<scalar_t, 128, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v2<scalar_t, 256, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                    case 8:
                        switch (S) {
                            case 64:
                                k_l2_interp_v2<scalar_t, 64, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v2<scalar_t, 128, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v2<scalar_t, 256, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                }
            });
            return out;
        }

        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {
            const auto saved = ctx->get_saved_variables();
            const Tensor coarse_grid = saved[0];
            const Tensor atoms = saved[1];

            const Tensor points = ctx->saved_data["points"].toTensor();
            const int64_t coarse_reso = ctx->saved_data["coarse_reso"].toInt();
            const int64_t fine_reso = ctx->saved_data["fine_reso"].toInt();

            const Tensor grad_output = grad_outputs[0];

            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();

            Tensor d_coarse_grid = torch::zeros_like(coarse_grid);
            Tensor d_atoms = torch::zeros_like(atoms);

            const dim3 grid_size_dcg(n_blocks_linear(points.size(0), CUDA_THREADS_PER_BLOCK / WARP_SIZE));
            const dim3 block_size_dcg(CUDA_THREADS_PER_BLOCK);

            const dim3 grid_size_da(points.size(0));
            const dim3 block_size_da(round_up(grad_output.size(1), 32), 8);  // D * S

            return {d_coarse_grid, d_atoms, Tensor(), Tensor(), Tensor(), Tensor(), Tensor()};
        }
};



Tensor l2_interp_v1(const Tensor &coarse_grid, const Tensor &atoms, const Tensor &points, const int64_t fine_reso,
                    const int64_t coarse_reso, const double fine_vl, const double coarse_vl)
{
    return L2InterpFunctionv1::apply(coarse_grid, atoms, points, fine_reso, coarse_reso);
}


Tensor l2_interp_v2(const Tensor &coarse_grid, const Tensor &atoms, const Tensor &points, const int64_t fine_reso,
                    const int64_t coarse_reso, const double fine_vl, const double coarse_vl)
{
    return L2InterpFunctionv2::apply(coarse_grid, atoms, points, fine_reso, coarse_reso);
}

static auto registry = torch::RegisterOperators()
                        .op("plenoxels::l2_interp_v2", &l2_interp_v2)
                        .op("plenoxels::l2_interp_v1", &l2_interp_v1);
