#include "hip/hip_runtime.h"
#include <cmath>
#include <stdexcept>
#include <tuple>

#include <torch/torch.h>
#include <torch/extension.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAGuard.h>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_reduce.cuh>

#include "cuda_util.cuh"


template <typename T, size_t N>
using Acc32 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int32_t>;
template <typename T, size_t N>
using Acc64 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int64_t>;

const int CUDA_THREADS_PER_BLOCK = 512;
const int WARP_SIZE = 32;
const int CUDA_WARPS_PER_BLOCK = CUDA_THREADS_PER_BLOCK / WARP_SIZE;


constexpr uint32_t n_blocks_linear(uint32_t n_elements, uint32_t n_threads_linear) {
    return (uint32_t)(n_elements + n_threads_linear - 1) / n_threads_linear;
}


__device__ __inline__ int32_t coo2idx(int32_t x, int32_t y, int32_t z, uint32_t grid_size) {
    return x + y * grid_size + z * grid_size * grid_size;
}

__constant__
static const float OFFSET[8][3] = {{-0.5, -0.5, -0.5}, {-0.5, -0.5, 0.5}, {-0.5, 0.5, -0.5}, {-0.5, 0.5, 0.5},
                                   {0.5, -0.5, -0.5},  {0.5, -0.5, 0.5},  {0.5, 0.5, -0.5},  {0.5, 0.5, 0.5}};

template<class scalar_t>
__device__ __inline__ void
calc_interp_weights(scalar_t * __restrict__ weights_out,
                    const scalar_t * __restrict__ point,
                    scalar_t * __restrict__ scratch)
{
    // Interpolation weight for fine coordinates to the center of top-left cell.
    scratch[0] = point[0] - 0.5f - myfloor(point[0] - 0.5f);
    scratch[1] = point[1] - 0.5f - myfloor(point[1] - 0.5f);
    scratch[2] = point[2] - 0.5f - myfloor(point[2] - 0.5f);
    weights_out[7] = scratch[0]         * scratch[1]         * scratch[2];
    weights_out[6] = scratch[0]         * scratch[1]         * (1.0 - scratch[2]);
    weights_out[5] = scratch[0]         * (1.0 - scratch[1]) * scratch[2];
    weights_out[4] = scratch[0]         * (1.0 - scratch[1]) * (1.0 - scratch[2]);
    weights_out[3] = (1.0 - scratch[0]) * scratch[1]         * scratch[2];
    weights_out[2] = (1.0 - scratch[0]) * scratch[1]         * (1.0 - scratch[2]);
    weights_out[1] = (1.0 - scratch[0]) * (1.0 - scratch[1]) * scratch[2];
    weights_out[0] = (1.0 - scratch[0]) * (1.0 - scratch[1]) * (1.0 - scratch[2]);
}


#define FWD_BLOCK_SIZE_X 32
#define FWD_BLOCK_SIZE_Y 16
#define NUM_POINTS_PER_THREAD 16
#define INNER_POINTS_PER_THREAD 4


template<class scalar_t, int32_t S, int32_t POW2_RF>
__global__ void
__launch_bounds__(FWD_BLOCK_SIZE_X * FWD_BLOCK_SIZE_Y, 2)
k_l2_interp_v2(Acc32<scalar_t, 2> coarse_grid,  // Rc^3, S
               Acc32<scalar_t, 3> atoms,        // Rf^3, D, S
               Acc32<scalar_t, 2> points,       // N, 3
               Acc32<scalar_t, 2> out,          // D, N
               const int32_t coarse_reso)
{
    /*
     * Thread-blocks are of size FWD_BLOCK_SIZE_X, FWD_BLOCK_SIZE_Y
     * The block will handle only FWD_BLOCK_SIZE_Y different fine-resolution points,
     * loading them all cooperatively into a_sh.
     *
     * The thread-block x-dimension is used to subdivide on the S dimension (num-atoms), to coalesce global loads.
     * Each row handles NUM_POINTS_PER_THREAD points in a loop. 
     *
     * The grid is of size:
     *  - num_points / FWD_BLOCK_SIZE_Y / NUM_POINTS_PER_THREAD
     *  - D
     *  - fine_reso^3 / FWD_BLOCK_SIZE_Y  TODO: This should be rounded up
     */
    constexpr int32_t num_s_per_thread = S / FWD_BLOCK_SIZE_X;
    constexpr int32_t fine_reso = 2 << (POW2_RF - 1);
    constexpr int32_t max_rf = fine_reso * fine_reso * fine_reso;
    typedef hipcub::WarpReduce<scalar_t> WarpReduce;

    const int32_t start_point_id = blockIdx.x * FWD_BLOCK_SIZE_Y * NUM_POINTS_PER_THREAD + threadIdx.y * NUM_POINTS_PER_THREAD;

    const int32_t rf_id = min(blockIdx.z * FWD_BLOCK_SIZE_Y + threadIdx.y, max_rf);
    const int32_t num_valid_rf_positions = min(FWD_BLOCK_SIZE_Y, max_rf - blockIdx.z * FWD_BLOCK_SIZE_Y);
    const int32_t dim_id = blockIdx.y;
    const int32_t s_idx = threadIdx.x;

    //int32_t cn[3];           // corner of coarse-neighbor cell in 'coarse-grid' coordinates
    scalar_t fn[3];           // corner of fine-neighbor cell in 'full-grid' coordinates
    //int32_t rfn[3];          // corner of fine-neighbor cell in 'full-grid' coordinates

    __shared__ scalar_t a_sh[FWD_BLOCK_SIZE_Y][S];
    __shared__ typename WarpReduce::TempStorage cub_storage[FWD_BLOCK_SIZE_Y];

    scalar_t accs[INNER_POINTS_PER_THREAD];
    scalar_t interpolation_weight;
    scalar_t cg_reg;

    for (int s = 0; s < num_s_per_thread; s++) {
        if (s * FWD_BLOCK_SIZE_X + s_idx < S) {
            a_sh[threadIdx.y][s * FWD_BLOCK_SIZE_X + s_idx] = atoms[rf_id][dim_id][s * FWD_BLOCK_SIZE_X + s_idx];
        } else {
            a_sh[threadIdx.y][s * FWD_BLOCK_SIZE_X + s_idx] = 0.0f;
        }
    }
    __syncthreads();

    int32_t cn_realcoo, fn_realcoo;
    int32_t cn_tmp, fn_tmp;
    for (int p = 0; p < NUM_POINTS_PER_THREAD; p += INNER_POINTS_PER_THREAD) {
        int j = 0;
        for (; j < INNER_POINTS_PER_THREAD; j++) {
            int point_id = start_point_id + p * INNER_POINTS_PER_THREAD + j;
            if (point_id >= out.size(1)) break;
            accs[j] = 0.0f;
            scalar_t fp[3] = {points[point_id][0] * fine_reso * coarse_reso, points[point_id][1] * fine_reso * coarse_reso, points[point_id][2] * fine_reso * coarse_reso};
            for (int i = 0; i < 8; i++) {
                fn[0] = clamp(floor2int(fp[0] + OFFSET[i][0]), 0, fine_reso * coarse_reso - 1);
                //fast_divmod_pow2<POW2_RF>(fn[0], cn[0], rfn[0]);
                fast_divmod_pow2<POW2_RF>(fn[0], cn_realcoo, fn_realcoo);
                fn[1] = clamp(floor2int(fp[1] + OFFSET[i][1]), 0, fine_reso * coarse_reso - 1);
                //fast_divmod_pow2<POW2_RF>(fn[1], cn[1], rfn[1]);
                fast_divmod_pow2<POW2_RF>(fn[1], cn_tmp, fn_tmp);
                cn_realcoo += cn_tmp * coarse_reso;
                fn_realcoo += fn_tmp * fine_reso;
                fn[2] = clamp(floor2int(fp[2] + OFFSET[i][2]), 0, fine_reso * coarse_reso - 1);
                //fast_divmod_pow2<POW2_RF>(fn[2], cn[2], rfn[2]);
                fast_divmod_pow2<POW2_RF>(fn[2], cn_tmp, fn_tmp);
                cn_realcoo += cn_tmp * coarse_reso * coarse_reso;
                fn_realcoo += fn_tmp * fine_reso * fine_reso - blockIdx.z * FWD_BLOCK_SIZE_Y;
                //cn_realcoo = coo2idx(cn[0], cn[1], cn[2], coarse_reso);
                //fn_realcoo = coo2idx(rfn[0], rfn[1], rfn[2], fine_reso) - blockIdx.z * FWD_BLOCK_SIZE_Y;

                if (fn_realcoo >= 0 && fn_realcoo < num_valid_rf_positions) {
                    interpolation_weight = (1.0f - myabs(fp[0] - static_cast<scalar_t>(fn[0]) - 0.5f)) *
                                           (1.0f - myabs(fp[1] - static_cast<scalar_t>(fn[1]) - 0.5f)) *
                                           (1.0f - myabs(fp[2] - static_cast<scalar_t>(fn[2]) - 0.5f));
                    for (int s = 0; s < num_s_per_thread; s++) {
                        // out-of-bounds reads will be zeroed out when multiplied by a_sh
                        cg_reg = coarse_grid[cn_realcoo][min(s * FWD_BLOCK_SIZE_X + s_idx, S)]; 
                        cg_reg *= interpolation_weight;
                        accs[j] = myfma(cg_reg, a_sh[fn_realcoo][s * FWD_BLOCK_SIZE_X + s_idx], accs[j]);
                    }
                }
            }
            accs[j] = WarpReduce(cub_storage[threadIdx.y]).Sum(accs[j]);
        }
        if (s_idx == 0) {
            for (int k = 0; k < j; k++) {
                atomicAdd(&out[dim_id][start_point_id + p * INNER_POINTS_PER_THREAD + k], accs[k]);
                //atomicAdd(&out[dim_id][blockIdx.x * FWD_BLOCK_SIZE_Y * NUM_POINTS_PER_THREAD + threadIdx.y * NUM_POINTS_PER_THREAD + p * INNER_POINTS_PER_THREAD + k], accs[k]);
            }
        }
    }
}




using torch::autograd::variable_list;
using torch::autograd::tensor_list;
using torch::autograd::Function;
using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::Tensor;


class L2InterpFunctionv2 : public Function<L2InterpFunctionv2> {
    public:
        static Tensor forward(AutogradContext *ctx,
                              Tensor coarse_grid,   // Rc^3, S
                              Tensor atoms,         // Rf^3, D, S
                              Tensor points,        // N, 3
                              int64_t fine_reso,
                              int64_t coarse_reso)
        {
            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();
            // Size checks
            if (coarse_grid.size(0) != coarse_reso * coarse_reso * coarse_reso) {
                throw std::invalid_argument("Coarse-grid has wrong first dimension");
            }
            if (coarse_grid.size(1) != atoms.size(2)) {
                throw std::invalid_argument("Coarse-grid and atoms dimension 1 doesn't match");
            }
            if (atoms.size(0) != fine_reso * fine_reso * fine_reso) {
                throw std::invalid_argument("Atoms has wrong first dimension");
            }
//            if (atoms.size(1) > 32) {
//                throw std::invalid_argument("Data dimension must be at most 32");
//            }

            ctx->save_for_backward({coarse_grid, atoms});
            ctx->saved_data["points"] = points;
            ctx->saved_data["fine_reso"] = fine_reso;
            ctx->saved_data["coarse_reso"] = coarse_reso;

            const int64_t D = atoms.size(1);
            const int64_t S = atoms.size(2);
            auto out = torch::zeros({D, points.size(0)}, atoms.options());

            const dim3 grid_size((uint32_t)(points.size(0) / FWD_BLOCK_SIZE_Y / NUM_POINTS_PER_THREAD), 
                                 (uint32_t)D, 
                                 (uint32_t)(fine_reso * fine_reso * fine_reso) / FWD_BLOCK_SIZE_Y);
            const dim3 block_size(FWD_BLOCK_SIZE_X, FWD_BLOCK_SIZE_Y);
//            const dim3 grid_size(n_blocks_linear(points.size(0), CUDA_WARPS_PER_BLOCK));
//            const dim3 block_size(CUDA_THREADS_PER_BLOCK);
//            const uint32_t shared_mem = CUDA_WARPS_PER_BLOCK * coarse_grid.size(1);

            fast_divmod fast_divmod_fine_reso((int32_t)fine_reso);
            AT_DISPATCH_FLOATING_TYPES(coarse_grid.scalar_type(), "dispatch_l2interpv2_fwd", [&] {
                switch(fine_reso) {
                    case 4:
                        switch (S) {
                            case 64:
                                k_l2_interp_v2<scalar_t, 64, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v2<scalar_t, 128, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v2<scalar_t, 256, 1><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                    case 8:
                        switch (S) {
                            case 64:
                                k_l2_interp_v2<scalar_t, 64, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 128:
                                k_l2_interp_v2<scalar_t, 128, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                            case 256:
                                k_l2_interp_v2<scalar_t, 256, 3><<<grid_size, block_size, 0, stream.stream()>>>(
                                    coarse_grid.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                                    points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                    (uint32_t)coarse_reso
                                );
                                break;
                        }
                        break;
                }
            });
            return out;
        }

        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {
            const auto saved = ctx->get_saved_variables();
            const Tensor coarse_grid = saved[0];
            const Tensor atoms = saved[1];

            const Tensor points = ctx->saved_data["points"].toTensor();
            const int64_t coarse_reso = ctx->saved_data["coarse_reso"].toInt();
            const int64_t fine_reso = ctx->saved_data["fine_reso"].toInt();

            const Tensor grad_output = grad_outputs[0];

            const at::cuda::CUDAGuard device_guard(coarse_grid.device());
            const auto stream = at::cuda::getCurrentCUDAStream();

            Tensor d_coarse_grid = torch::zeros_like(coarse_grid);
            Tensor d_atoms = torch::zeros_like(atoms);

            const dim3 grid_size_dcg(n_blocks_linear(points.size(0), CUDA_THREADS_PER_BLOCK / WARP_SIZE));
            const dim3 block_size_dcg(CUDA_THREADS_PER_BLOCK);

            const dim3 grid_size_da(points.size(0));
            const dim3 block_size_da(round_up(grad_output.size(1), 32), 8);  // D * S

            return {d_coarse_grid, d_atoms, Tensor(), Tensor(), Tensor(), Tensor(), Tensor()};
        }
};


Tensor l2_interp_v2(const Tensor &coarse_grid, const Tensor &atoms, const Tensor &points, const int64_t fine_reso,
                    const int64_t coarse_reso, const double fine_vl, const double coarse_vl)
{
    return L2InterpFunctionv2::apply(coarse_grid, atoms, points, fine_reso, coarse_reso);
}

static auto registry = torch::RegisterOperators()
                        .op("plenoxels::l2_interp_v2", &l2_interp_v2);
