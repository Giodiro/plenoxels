#include "hip/hip_runtime.h"
#include <cmath>
#include <tuple>

#include <torch/torch.h>
#include <torch/extension.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAGuard.h>
#include <cub/warp/warp_reduce.cuh>

template <typename T, size_t N>
using Acc32 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int32_t>;
template <typename T, size_t N>
using Acc64 = torch::GenericPackedTensorAccessor<T, N, torch::RestrictPtrTraits, int64_t>;


constexpr uint32_t n_blocks_linear(uint32_t n_elements, uint32_t n_threads_linear) {
    return (uint32_t)(n_elements + n_threads_linear - 1) / n_threads_linear;
}

__host__ __device__ __forceinline__ float myfma(float a, float b, float c) { return fmaf(a, b, c); }
__host__ __device__ __forceinline__ double myfma(double a, double b, double c) { return fma(a, b, c); }
__host__ __device__ __forceinline__ float myfloor(float a) { return floorf(a); }
__host__ __device__ __forceinline__ double myfloor(double a) { return floor(a); }
/*
 * Linear interpolation
 * implements (1 - w) * a + w * b via a subtraction and a fused multiply-add.
 * TODO: This only works for floats due to use of fmaf.
 */
template<typename T>
__host__ __device__ __inline__ T lerp(T a, T b, T w) {
    return myfma(w, b - a, a);
}

template<typename index_t, typename data_t>
__device__ __inline__ float trilerp_one(const index_t * __restrict__ n_idx,
                                        const float   * __restrict__ pos,
                                        const data_t  * __restrict__ data,
                                        const uint32_t stride,
                                        const uint32_t grid_size,
                                        const uint32_t data_idx)
{
    // data: [x][y][z][sh-data]
    const uint32_t offz = stride;            // stride=stride
    const uint32_t offy = grid_size * offz;  // stride=stride * grid_size
    const uint32_t offx = grid_size * offy;  // stride=stride * grid_size * grid_size

    const data_t * __restrict__ data_ptr = data + offx * n_idx[0] +
                                                  offy * n_idx[1] +
                                                  offz * n_idx[2] +
                                                  data_idx;

    const float ix0y0 = lerp(data_ptr[0], data_ptr[offz], pos[2]);            // (1-z) * (x,y,z) + (z) * (x,y,z+1)
    const float ix0y1 = lerp(data_ptr[offy], data_ptr[offy + offz], pos[2]);  // (1-z) * (x,y+1,z) + (z) * (x,y+1,z+1)
    const float ix0 = lerp(ix0y0, ix0y1, pos[1]);                             // (1-y) * ix0y0 + (y) * ix0y1
    const float ix1y0 = lerp(data_ptr[offx], data_ptr[offx + offz], pos[2]);  // (1-z) * (x+1,y,z) + (z) * (x+1,y,z+1)
    const float ix1y1 = lerp(data_ptr[offy + offx], data_ptr[offy + offx + offz], pos[2]);  // (1-z)*(x+1,y+1,z)+z*(x+1,y+1,z+1)
    const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
    return lerp(ix0, ix1, pos[0]);
}

template<typename data_t, typename out_t>
__device__ __inline__ out_t trilerp_precomputed(const out_t   * __restrict__ pos,
                                                const data_t  * __restrict__ data)
{
    const out_t ix0y0 = lerp(static_cast<out_t>(data[0]), static_cast<out_t>(data[1]), pos[0]);
    const out_t ix0y1 = lerp(static_cast<out_t>(data[2]), static_cast<out_t>(data[3]), pos[0]);
    const out_t ix0 = lerp(ix0y0, ix0y1, pos[1]);
    const out_t ix1y0 = lerp(static_cast<out_t>(data[4]), static_cast<out_t>(data[5]), pos[0]);
    const out_t ix1y1 = lerp(static_cast<out_t>(data[6]), static_cast<out_t>(data[7]), pos[0]);
    const out_t ix1 = lerp(ix1y0, ix1y1, pos[1]);
    return lerp(ix0, ix1, pos[2]);
}

template<typename out_t>
__device__ __inline__ void unnormalize_pos(out_t * __restrict__ pos,
                                           int32_t * __restrict__ idx,
                                           const uint32_t grid_size)
{
    #pragma unroll 3
    for (int j = 0; j < 3; j++) {  // this work is repeated unnecessarily for all threads in warp.
        pos[j] = pos[j] * grid_size - 0.5;
        pos[j] = min(static_cast<out_t>(grid_size - 1), max(pos[j], 0.0));
        idx[j] = static_cast<int32_t>(myfloor(pos[j]));
        //pos[j] = pos[j] * (grid_size - 1);
        //pos[j] = min(max(pos[j], 0.0f), static_cast<out_t>(grid_size - 1));
        //n_idx[j] = min(static_cast<int32_t>(pos[j]), grid_size - 2);
        pos[j] -= static_cast<out_t>(idx[j]);
    }
}


template<typename query_t, typename sh_t, typename out_t>
__global__ void k_l2_interp(Acc64<query_t, 2> Q,       // N x S
                            Acc32<sh_t, 3> A,          // S x R^3 x D
                            Acc32<out_t, 2> O,         // N x D
                            Acc32<out_t, 2> positions,  // N x 3
                            const uint32_t grid_size
                           )
{
    const uint32_t point_id = blockIdx.x * (blockDim.x / 32) + threadIdx.x / 32;
    const uint32_t warp_lane = threadIdx.x % 32;
    const uint32_t S = A.size(0);
    const uint32_t D = A.size(2);
    if (warp_lane >= D || point_id >= Q.size(0)) { return; }
    //printf("blockIdx.x %d, threadIdx.x %d, point %d\n", blockIdx.x, threadIdx.x, point_id);
    out_t pos[3] = {positions[point_id][0], positions[point_id][1], positions[point_id][2]};
    int32_t n_idx[3];
    unnormalize_pos(pos, n_idx, grid_size);
    sh_t neighbor_data[8] = {0.};
    const uint32_t offx = 1;                 // stride=stride
    const uint32_t offy = offx * grid_size;  // stride=stride * grid_size
    const uint32_t offz = offy * grid_size;  // stride=stride * grid_size * grid_size
    const uint32_t offdata = offx * n_idx[0] + offy * n_idx[1] + offz * n_idx[2];
    for (int s = 0; s < S; s++) {
        // Load s-th weight from global
        const sh_t weight = static_cast<sh_t>(Q[point_id][s]);
        neighbor_data[0] = myfma(weight, A[s][offdata][warp_lane], neighbor_data[0]);
        neighbor_data[1] = myfma(weight, A[s][offdata + offx][warp_lane], neighbor_data[1]);
        neighbor_data[2] = myfma(weight, A[s][offdata + offy][warp_lane], neighbor_data[2]);
        neighbor_data[3] = myfma(weight, A[s][offdata + offy + offx][warp_lane], neighbor_data[3]);
        neighbor_data[4] = myfma(weight, A[s][offdata + offz][warp_lane], neighbor_data[4]);
        neighbor_data[5] = myfma(weight, A[s][offdata + offz + offx][warp_lane], neighbor_data[5]);
        neighbor_data[6] = myfma(weight, A[s][offdata + offz + offy][warp_lane], neighbor_data[6]);
        neighbor_data[7] = myfma(weight, A[s][offdata + offz + offy + offx][warp_lane], neighbor_data[7]);
    }
    O[point_id][warp_lane] = trilerp_precomputed(pos, neighbor_data);
}

template<typename query_t, typename sh_t, typename out_t>
__global__ void k_l2_interp_dA(Acc32<out_t, 2> grad_output,  // N x D
                               Acc32<sh_t, 3> DA,    // S x R^3 x D
                               Acc32<out_t, 2> positions,
                               Acc32<query_t, 2> Q,
                               const uint32_t grid_size
                               )
{
    const uint32_t atom_id = blockIdx.x * (blockDim.x / 32) + threadIdx.x / 32;
    const uint32_t point_id = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t warp_lane = threadIdx.x % 32;
    const uint32_t S = DA.size(0);
    const uint32_t D = DA.size(2);
    const uint32_t N = positions.size(0);
    if (warp_lane >= D || atom_id >= S || point_id >= N) { return; }

    const uint32_t offx = 1;                 // stride=stride
    const uint32_t offy = offx * grid_size;  // stride=stride * grid_size
    const uint32_t offz = offy * grid_size;  // stride=stride * grid_size * grid_size

    out_t pos[3];
    int32_t n_idx[3];
    pos[0] = positions[point_id][0];
    pos[1] = positions[point_id][1];
    pos[2] = positions[point_id][2];
    unnormalize_pos(pos, n_idx, grid_size);
    const uint32_t offdata = offx * n_idx[0] + offy * n_idx[1] + offz * n_idx[2];
    const out_t ax = 1.f - pos[0];
    const out_t ay = 1.f - pos[1];
    const out_t az = 1.f - pos[2];
    const out_t go = grad_output[point_id][warp_lane];
    const sh_t weight = static_cast<sh_t>(Q[point_id][atom_id]) * go;
    DA[atom_id][offdata][warp_lane] += static_cast<sh_t>(ax * ay * az * weight);
    DA[atom_id][offdata + offx][warp_lane] += static_cast<sh_t>(az * ay * pos[0] * weight);
    DA[atom_id][offdata + offy][warp_lane] += static_cast<sh_t>(az * pos[1] * ax * weight);
    DA[atom_id][offdata + offy + offx][warp_lane] += static_cast<sh_t>(az * pos[1] * pos[0] * weight);
    DA[atom_id][offdata + offz][warp_lane] += static_cast<sh_t>(pos[2] * ay * ax * weight);
    DA[atom_id][offdata + offz + offx][warp_lane] += static_cast<sh_t>(pos[2] * ay * pos[0] * weight);
    DA[atom_id][offdata + offz + offy][warp_lane] += static_cast<sh_t>(pos[2] * pos[1] * ax * weight);
    DA[atom_id][offdata + offz + offy + offx][warp_lane] += static_cast<sh_t>(pos[2] * pos[1] * pos[0] * weight);
}


template<typename query_t, typename sh_t, typename out_t>
__global__ void k_l2_interp_dQ(Acc32<out_t, 2> grad_output,  // N x D
                               Acc32<query_t, 2> DQ,  // N x S
                               Acc32<out_t, 2> positions,
                               Acc32<sh_t, 3> A,
                               const uint32_t grid_size
                               )
{
    const uint32_t point_id = blockIdx.x * (blockDim.x / 32) + threadIdx.x / 32;
    const uint32_t warp_lane = threadIdx.x % 32;
    const uint32_t S = A.size(0);
    const uint32_t D = A.size(2);
    if (warp_lane >= D || point_id >= DQ.size(0)) { return; }
    __shared__ typename hipcub::WarpReduce<out_t>::TempStorage temp_storage;

    out_t pos[3] = {positions[point_id][0], positions[point_id][1], positions[point_id][2]};
    int32_t n_idx[3];
    unnormalize_pos(pos, n_idx, grid_size);
    const out_t ax = 1.f - pos[0];
    const out_t ay = 1.f - pos[1];
    const out_t az = 1.f - pos[2];
    const uint32_t offx = A.stride(1);                 // stride=stride
    const uint32_t offy = offx * grid_size;  // stride=stride * grid_size
    const uint32_t offz = offy * grid_size;  // stride=stride * grid_size * grid_size
    uint32_t A_offset = offx * n_idx[0] + offy * n_idx[1] + offz * n_idx[2] + warp_lane;

    sh_t* __restrict__  A_ptr = A.data();
    const uint32_t A_stride0 = A.stride(0);

    const out_t go = grad_output[point_id][warp_lane];
    out_t dq_temp;
    for (int s = 0; s < S; s++) {
        // Gradient with respect to atoms (DA) is summed over all points
        // Gradient with respect to queries (DQ) is summed over all dimensions (warp lanes)
        dq_temp =       ax * ay * az *            A_ptr[A_offset];
        dq_temp = myfma(az * ay * pos[0],         A_ptr[A_offset + offx],               dq_temp);
        dq_temp = myfma(az * pos[1] * ax,         A_ptr[A_offset + offy],               dq_temp);
        dq_temp = myfma(az * pos[1] * pos[0],     A_ptr[A_offset + offy + offx],        dq_temp);
        dq_temp = myfma(pos[2] * ay * ax,         A_ptr[A_offset + offz],               dq_temp);
        dq_temp = myfma(pos[2] * ay * pos[0],     A_ptr[A_offset + offz + offx],        dq_temp);
        dq_temp = myfma(pos[2] * pos[1] * ax,     A_ptr[A_offset + offz + offy],        dq_temp);
        dq_temp = myfma(pos[2] * pos[1] * pos[0], A_ptr[A_offset + offx + offy + offz], dq_temp);
        dq_temp *= go;
        dq_temp = hipcub::WarpReduce<out_t>(temp_storage).Sum(dq_temp, D);
        if (warp_lane == 0) {
            DQ[point_id][s] = static_cast<query_t>(dq_temp);
        }
        A_offset += A_stride0;
    }
}

template<typename query_t, typename sh_t, typename out_t>
__global__ void k_l2_interp_bwd(Acc32<out_t, 2> grad_output,  // N x D
                                Acc64<query_t, 2> DQ,  // N x S
                                Acc32<sh_t, 3> DA,    // S x R^3 x D
                                Acc32<out_t, 2> positions,
                                Acc32<query_t, 2> Q,
                                Acc32<sh_t, 3> A,
                                const uint32_t grid_size
                                )
{
    const uint32_t point_id = blockIdx.x * (blockDim.x / 32) + threadIdx.x / 32;
    const uint32_t warp_lane = threadIdx.x % 32;
    const uint32_t S = A.size(0);
    const uint32_t D = A.size(2);
    if (warp_lane >= D || point_id >= Q.size(0)) { return; }
    __shared__ typename hipcub::WarpReduce<out_t>::TempStorage temp_storage;

    out_t pos[3] = {positions[point_id][0], positions[point_id][1], positions[point_id][2]};
    int32_t n_idx[3];
    unnormalize_pos(pos, n_idx, grid_size);
    const out_t ax = 1.f - pos[0];
    const out_t ay = 1.f - pos[1];
    const out_t az = 1.f - pos[2];
    const uint32_t offx = A.stride(1);                 // stride=stride
    const uint32_t offy = offx * grid_size;  // stride=stride * grid_size
    const uint32_t offz = offy * grid_size;  // stride=stride * grid_size * grid_size
    uint32_t A_offset = offx * n_idx[0] + offy * n_idx[1] + offz * n_idx[2] + warp_lane;

    sh_t* __restrict__  A_ptr = A.data();
    sh_t* __restrict__ DA_ptr = DA.data();
    const uint32_t A_stride0 = A.stride(0);

    const out_t go = grad_output[point_id][warp_lane];
    out_t iw;     // interpolation weight
    uint32_t il;  // interpolation location within 2nd level grid
    out_t dq_temp;
    for (int s = 0; s < S; s++) {
        // Gradient with respect to atoms (DA) is summed over all points
        // Gradient with respect to queries (DQ) is summed over all dimensions (warp lanes)
        const out_t weight = static_cast<out_t>(Q[point_id][s]) * go;
        iw = ax * ay * az;
        il = A_offset;
        dq_temp = iw * A_ptr[il];
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = az * ay * pos[0];
        il = A_offset + offx;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = az * pos[1] * ax;
        il = A_offset + offy;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = az * pos[1] * pos[0];
        il = A_offset + offy + offx;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = pos[2] * ay * ax;
        il = A_offset + offz;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = pos[2] * ay * pos[0];
        il = A_offset + offz + offx;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = pos[2] * pos[1] * ax;
        il = A_offset + offz + offy;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        iw = pos[2] * pos[1] * pos[0];
        il = A_offset + offx + offy + offz;
        dq_temp = myfma(iw, A_ptr[il], dq_temp);
        atomicAdd(&DA_ptr[il], static_cast<sh_t>(iw * weight));

        dq_temp *= go;
        dq_temp = hipcub::WarpReduce<out_t>(temp_storage).Sum(dq_temp);
        if (warp_lane == 0) {
            DQ[point_id][s] = static_cast<query_t>(dq_temp);
        }
        A_offset += A_stride0;
    }
}

/*
 * PyTorch Wrappers
 */


using torch::autograd::variable_list;
using torch::autograd::tensor_list;
using torch::autograd::Function;
using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::Tensor;


class L2InterpFunction : public Function<L2InterpFunction> {
    public:
        static Tensor forward(AutogradContext *ctx,
                              Tensor queries,
                              Tensor atoms,
                              Tensor points)
        {
            const at::cuda::CUDAGuard device_guard(queries.device());
            const auto stream = at::cuda::getCurrentCUDAStream();
            ctx->save_for_backward({queries, atoms});
            ctx->saved_data["points"] = points;


            const uint32_t l2_grid_size = (uint32_t)std::cbrt(atoms.size(1));
            auto out = torch::zeros({queries.size(0), atoms.size(2)}, torch::dtype(queries.dtype()).device(queries.device()));
            const uint32_t threads_per_block = 512;
            AT_DISPATCH_FLOATING_TYPES(queries.scalar_type(), "dispatch_l2interp_fwd", [&] {
                k_l2_interp<scalar_t, scalar_t, scalar_t>
                    <<< n_blocks_linear(queries.size(0), threads_per_block / 32), threads_per_block, 0, stream.stream()>>>
                    (queries.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                     atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                     out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     l2_grid_size);
            });
            return out;
        }
        static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs)
        {
            const auto saved = ctx->get_saved_variables();
            const Tensor queries = saved[0];
            const Tensor atoms = saved[1];
            const Tensor points = ctx->saved_data["points"].toTensor();
            const Tensor grad_output = grad_outputs[0];

            const at::cuda::CUDAGuard device_guard(queries.device());
            const auto stream = at::cuda::getCurrentCUDAStream();

            const uint32_t l2_grid_size = (uint32_t)std::cbrt(atoms.size(1));
            Tensor d_queries = torch::zeros_like(queries);
            Tensor d_atoms = torch::zeros_like(atoms);
            const uint32_t threads_per_block = 512;
            AT_DISPATCH_FLOATING_TYPES(queries.scalar_type(), "dispatch_l2interp_bwd", [&] {
                k_l2_interp_bwd<scalar_t, scalar_t, scalar_t>
                    <<< n_blocks_linear(queries.size(0), threads_per_block / 32), threads_per_block, 0, stream.stream()>>>
                    (grad_output.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                     d_queries.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     d_atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                     points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     queries.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                     l2_grid_size);
/*                k_l2_interp_dA<scalar_t, scalar_t, scalar_t>
                    <<< n_blocks_linear(atoms.size(0), 32 / 32), 32, 0, stream.stream()>>>
                    (grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     d_atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                     points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     queries.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     l2_grid_size);
                k_l2_interp_dQ<scalar_t, scalar_t, scalar_t>
                    <<< n_blocks_linear(queries.size(0), threads_per_block / 32), threads_per_block, 0, stream.stream()>>>
                    (grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     d_queries.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     points.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                     atoms.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                     l2_grid_size);*/
            });
            return {d_queries, d_atoms, Tensor()};
        }
};


Tensor l2_interp(const Tensor &queries, const Tensor &atoms, const Tensor &points) 
{
    return L2InterpFunction::apply(queries, atoms, points);
}

static auto registry = torch::RegisterOperators()
                        .op("plenoxels::l2_interp", &l2_interp);

